#include "hip/hip_runtime.h"

#define blockSizeBubbleSort 1024
#define blockSizeDotProd 1024
#define isPrint false

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

/** Swaps elements idx and idx+1 if the element idx+1 is greater than the
 * 	element idx, if idx is lower than the current step by 2 and if idx is
 *  not n-1
 *  @param deviceArray Sorted array
 *	@param n Size of the sorted array
 * 	@param step Current step
 */ 
__global__ void bubbleSortKernel(float *deviceArray, int n, int step){
	// Thread linear id
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	float temp; // Temporary variable
	if (idx<(n-1)) {
    if ((step-2)>=idx){
      if (deviceArray[idx]<deviceArray[idx+1]){
        temp = deviceArray[idx];
        deviceArray[idx]=deviceArray[idx+1];
        deviceArray[idx+1] = temp;
      }
    }
  }
}

/** Sorts hostArray with the help of the Bubble sorting (CUDA is used)
 *	@param hostArray Sorted array
 *	@param n Size of the sorted array
 */
void bubbleSortCUDA(float *hostArray, int n){
	float *deviceArray; // declaration of a device copy of the sorted array
	hipMalloc(&deviceArray, n * sizeof(float)); // Memory allocation
	// Copy host vector to device memory
	hipMemcpy(deviceArray, hostArray, n*sizeof(float), hipMemcpyHostToDevice);
	int gridSize = n / blockSizeBubbleSort + 1; // Size of a CUDA-grid
	// Bubble sort loop
	for (int step = 0; step <= n+n; step++){
		bubbleSortKernel<<<gridSize, blockSizeBubbleSort>>>(deviceArray, n, step);
		hipDeviceSynchronize();
	}
	// Copy back to host
	hipMemcpy(hostArray, deviceArray, n*sizeof(float), hipMemcpyDeviceToHost);
	// Release memory
	hipFree(deviceArray);
}

/** Sorts hostArray with the help of the Bubble sorting (CUDA is not used)
 *	@param hostArray Sorted array
 *	@param n Size of the sorted array
 */
void bubbleSortCPU(float *hostArray, int n){
	float temp; // Temporary variable
	for (int i = 0; i < n; i++){
    for (int j = 0; j < n-i-1; j++) {
      if (hostArray[j]<hostArray[j+1]){
        temp = hostArray[j];
        hostArray[j] = hostArray[j+1];
        hostArray[j+1] = temp;
      }
    }
  }
}

/** Computes intermediate result of dot product
 *	@param blockSize Size of a CUDA-block
 *	@param deviceA First vector
 *	@param deviceB Second vector
 *	@param deviceC Intermediate result
 *	@param n Size of the first and the second vectors
 */
template<int blockSize>
__global__ void dotProdKernel(float const *deviceA, float const *deviceB, float *deviceC, int n){
	// shared array for cache (this is a shared array for the whole threads in the current block)
	__shared__ float cache[blockSize];
	// Get our thread linear ID
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float sum = 0;
  // Computation of sum of all elements that lie on this thread
  while (idx < n) {
  	sum += deviceA[idx] * deviceB[idx];
  	idx += blockDim.x * gridDim.x;
  }
  cache[threadIdx.x] = sum; // Filling cache
  __syncthreads(); // Barier synchronisation
  // Summation the cache (on the current block)
  int i = blockDim.x / 2;
  while (i != 0){
 		if (threadIdx.x < i) cache[threadIdx.x] += cache[threadIdx.x + i];
 		__syncthreads(); // Barier synchronisation
 		i /= 2;
 	}
 	// Sum of the cache on the current block is at cache[0] 
 	// Saving this result for each blocks
 	if (threadIdx.x == 0) deviceC[blockIdx.x] = cache[0];
}

/** Computes dot product (CUDA is used)
 *	@param hostA First array
 *	@param hostB Second array
 *	@param n Size of arrays
 */
float dotProdCUDA(float const *hostA, float const *hostB, int n){
	float *hostC; // declaration of vectors
	float *deviceA, *deviceB, *deviceC;
	// Number of thread blocks per grid
  int gridDimDotProd = (n + blockSizeDotProd - 1) / blockSizeDotProd;
  // Size in bytes of the vector C
  size_t bytesC = gridDimDotProd * sizeof(float);
	// Vectors allocation
	hostC = (float*)malloc(bytesC);
	hipMalloc(&deviceA, n * sizeof(float));
  hipMalloc(&deviceB, n * sizeof(float));
	hipMalloc(&deviceC, bytesC);	
	// Copy host vectors to device
  hipMemcpy(deviceA, hostA, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, n * sizeof(float), hipMemcpyHostToDevice);
  // Execute the kernel
  dotProdKernel<blockSizeDotProd><<<gridDimDotProd, blockSizeDotProd>>>(deviceA, deviceB, deviceC, n);
  // Copy array back to host
  hipMemcpy(hostC, deviceC, bytesC, hipMemcpyDeviceToHost);
  // Release device memory
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  // Finish up on the host
  float res = 0;
  for (int i = 0; i < gridDimDotProd; i++){
  	res += hostC[i];
  }
  return res;
}

/** Computes dot product (CUDA is not used)
 *	@param hostA First array
 *	@param hostB Second array
 *	@param n Size of arrays
 */
float dotProdCPU(float const *hostA, float const *hostB, int n){
	float res = 0;
	for (int i = 0; i < n; i++) {
		res += hostA[i] * hostB[i];
	}
	return res;
}

/** Prints array
 *	@param a Array
 *	@param n Array length
 */
void printArray(float *a, int n){
	printf("[%f", a[0]);
	for (int i = 1; i < n; i++) {
		printf(",%f", a[i]);
	}
	printf("]\n");
}

float taskCPU(int n){
	// Arrays declaration
	float *hostA, *hostB;
	clock_t t1, t2;
	// CPU variant
	if (isPrint) printf("CUDA is not used\n");
	t1 = clock();
	// Arrays allocation
	hostA = (float*)malloc(n*sizeof(float));
	hostB = (float*)malloc(n*sizeof(float));
	// Arrays initialization
	for (int i=0; i<n; i++){
		hostA[i] = (float)rand()/(float)(RAND_MAX);
		hostB[i] = (float)rand()/(float)(RAND_MAX);
	}
	// Print array
	if (isPrint) printf("A\t");
	if (isPrint) printArray(hostA, n);
	if (isPrint) printf("B\t");
	if (isPrint) printArray(hostB, n);
	// Sorting
	bubbleSortCPU(hostA, n);
	bubbleSortCPU(hostB, n);
	// Print arrays
	if (isPrint) printf("sorted A\t");
	if (isPrint) printArray(hostA, n);
	if (isPrint) printf("sorted B\t");
	if (isPrint) printArray(hostB, n);
	// Dot product
	float dot_prod = dotProdCPU(hostA, hostB, n);
	if (isPrint) printf("dot_prods\t%f\n", dot_prod);
	t2 = clock();
	// Release memory
	free(hostA);
	free(hostB);
	return (float)(t2-t1)/CLOCKS_PER_SEC;
}

float taskCUDA(int n){
	// Arrays declaration
	float *hostA, *hostB;
	clock_t t1, t2;
	// CPU variant
	if (isPrint) printf("CUDA is not used\n");
	t1 = clock();
	// Arrays allocation
	hostA = (float*)malloc(n*sizeof(float));
	hostB = (float*)malloc(n*sizeof(float));
	// Arrays initialization
	for (int i=0; i<n; i++){
		hostA[i] = (float)rand()/(float)(RAND_MAX);
		hostB[i] = (float)rand()/(float)(RAND_MAX);
	}
	// Print array
	if (isPrint) printf("A\t");
	if (isPrint) printArray(hostA, n);
	if (isPrint) printf("B\t");
	if (isPrint) printArray(hostB, n);
	// Sorting
	bubbleSortCUDA(hostA, n);
	bubbleSortCUDA(hostB, n);
	// Print arrays
	if (isPrint) printf("sorted A\t");
	if (isPrint) printArray(hostA, n);
	if (isPrint) printf("sorted B\t");
	if (isPrint) printArray(hostB, n);
	// Dot product
	float dot_prod = dotProdCUDA(hostA, hostB, n);
	if (isPrint) printf("dot_prods\t%f\n", dot_prod);
	t2 = clock();
	// Release memory
	free(hostA);
	free(hostB);
	return (float)(t2-t1)/CLOCKS_PER_SEC;
}

int main(int argc, char *argv[]){
	int n = atoi(argv[1]); // Arrays size
	//printf("%d %f %f\n", n, taskCPU(n), taskCUDA(n));
	printf("%d %f\n", n, taskCUDA(n));
	return 0;	
}





















